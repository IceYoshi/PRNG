#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <signal.h>
#include "util.hpp"
#include <unistd.h>

using namespace std;

unsigned int *random_numbers;
unsigned int *random_numbers_dev;

/* launch one block */
#define BLOCKSIZE 256
#define NUM_THREADS BLOCKSIZE
#define NUM_RANDOM_NUMBERS_DEV (1 << 14)

#include RANDOM_NUMBER_GENERATOR

enum { NUM_RANDOMS = NUM_RANDOM_NUMBERS_DEV };

void
initialize_cuda()
{
	choose_device();

	CUDA_CHECK_ERROR(hipHostMalloc(&random_numbers,
				sizeof(*random_numbers) * NUM_RANDOMS));

	CUDA_CHECK_ERROR(hipMalloc(&random_numbers_dev,
				sizeof(*random_numbers_dev) * NUM_RANDOMS));
}

__global__ void
kernel_generate_randoms(const RNG::DevParameters params, unsigned int *random_numbers)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	RNG::RNGState rng_state;
	RNG::initialize(&params, &rng_state);

	unsigned int rnds[RNG::num_randoms_per_call];

	for(int i = 0; i < NUM_RANDOMS; i += RNG::num_randoms_per_call) {
		RNG::generate_random_numbers(&rng_state, rnds, 1, RNG::num_randoms_per_call);
		if(idx == 0) {
			for(int j = 0; j < RNG::num_randoms_per_call; j++) {
				random_numbers[i + j] = rnds[j];
			}
		}
	}

	RNG::finalize(&params, &rng_state);
}

void
handle_sig_pipe(int sig)
{
	exit(EXIT_SUCCESS);
}

int
main(int argc, char **argv)
{
	if(isatty(1)) {
		cerr << "i won't write to a tty" << endl;
		exit(EXIT_FAILURE);
	}

	signal(SIGPIPE, handle_sig_pipe);

	initialize_cuda();

	RNG::DevParameters rng_parameters;
	RNG::initialize_rng(&rng_parameters);

	dim3 block(BLOCKSIZE, 1, 1);
	dim3 grid(NUM_THREADS / BLOCKSIZE, 1, 1);

	for(;;) {
		kernel_generate_randoms<<< grid, block >>> (rng_parameters, random_numbers_dev);
		CUDA_CHECK_ERROR(hipGetLastError());
		CUDA_CHECK_ERROR(hipMemcpy(random_numbers, random_numbers_dev,
					sizeof(*random_numbers_dev) * NUM_RANDOMS,
					hipMemcpyDeviceToHost));

		unsigned int *ptr = random_numbers;
		int s = 0, cnt = sizeof(*ptr) * NUM_RANDOMS;
		while((s = write(1, ptr, cnt)) < cnt) {
			if(s < 0) {
				perror("error writing");
				exit(EXIT_FAILURE);
			}
			ptr += s;
			cnt -= s;
		}
	}

	return 0;
}
